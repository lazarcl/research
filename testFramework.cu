#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <unistd.h>
#include <string>
#include "config.cpp"


template <class K>
class TestRunner {
public:


  //max number of times to attempt getting a good sample
  const int maxTestRuns = 9;

  //number of samples to igore from beg and end while analyzing data
  const int ignoreSampleCount = 80;
  
  //int deviceIDNum: GPU device to do all work/sampling on
  int deviceIDNum;

  //acceptable data error percentage in analysis
  float acceptableError;

  //device properties
  hipDeviceProp_t deviceProp;

  //desired filename for output
  const char *outputName; 

  //hold data from samples in vectors
  std::vector<float> powerData;
  std::vector<int> tempData;

  //the deviceID in nvml format
  nvmlDevice_t nvmlDeviceID;

  //the result code of the previous nvml call
  nvmlReturn_t nvmlResult;

  //CUDA timing events to clalculate test kernel's runtime
  hipEvent_t gpuStart, gpuStop, kernelFinished;

  //class that holds the kernel to run
  K *testClass;

  TestRunner(K *tester, const char *outputName, float acceptableError=0.03) 
          : outputName(outputName), acceptableError(acceptableError) {
    
    testClass = tester;

    if (!setDevice()) {
      printf("Could not find or set device. Check settings in 'config.cpp'\n");
      exit(1);
    }
    printf("  Device initialized to nvml slot: %d, and cuda name: '%s'\n", config_t.deviceID, config_t.deviceName);
    

    nvmlResult = nvmlInit();
    if ( nvmlResult != NVML_SUCCESS )
    {
      printf("NVML Init fail: %s\n", nvmlErrorString(nvmlResult));
      exit(1);
    }

    nvmlResult = nvmlDeviceGetHandleByIndex(deviceIDNum, &nvmlDeviceID);
    if (nvmlResult != NVML_SUCCESS) {
      printf("failed getting device handle by index: %s\n", nvmlErrorString(nvmlResult));
      exit(1);
    }
  }

bool setDevice() {
  deviceIDNum = config_t.deviceID;
  std::string desiredDeviceName = config_t.deviceName;

  int devicesCount;
  CUDA_ERROR( hipGetDeviceCount(&devicesCount) );
  
  for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
  {
      CUDA_ERROR( hipGetDeviceProperties(&deviceProp, deviceIndex) );
      if (std::string(deviceProp.name) == desiredDeviceName)
      {
	  printf("total const mem: %d\n",(int) deviceProp.totalConstMem);
          CUDA_ERROR( hipSetDevice(deviceIndex) );
          return true;
      }
      if (deviceIndex > 10) {
          return false;
      }
  }
  return false;
}

  /*
  start measurement of device's power/temp
    initial setup of nvml and vars
  */
  void setupSampling()
  {

    //reserve about 6 min of sampling space
    powerData.clear();
    powerData.reserve(750000);  
    tempData.clear();
    tempData.reserve(750000);  

    CUDA_ERROR( hipEventCreate(&gpuStart) );
    CUDA_ERROR( hipEventCreate(&gpuStop) );
    CUDA_ERROR( hipEventRecord(gpuStart) );
  }


  /*
  takeMeasurements
    loop that samples until kernel has finished
    return bool - true: good sample graph, false: bad samples
  */
  void runSampling() {
    unsigned int powerLevel, tempLevel;
    CUDA_ERROR( hipEventCreate(&kernelFinished) );
    CUDA_ERROR( hipEventRecord(kernelFinished) ); 

    while (hipEventQuery(kernelFinished) != hipSuccess) {
      powerLevel = getPowerUseage();
      tempLevel = getDeviceTemp();
      powerData.push_back(powerLevel);
      tempData.push_back(tempLevel);
    }
    CUDA_ERROR( hipEventRecord(gpuStop, 0) );
    CUDA_ERROR( hipEventSynchronize(gpuStop) );
  }


  /*
  get a good test reading
    keep running test until:
      good reading && temp was constant
      haven't looped too many times
    warn if test case is not long enough
    print each time test case is run
    if loop runs to maxItr, then print to file anyways
  */
  
  void getGoodSample() {
    unsigned int prevTemp = 0;
    unsigned int curTemp = getDeviceTemp();
    bool badSampleData = true;
    int curRun = 1;

    testClass->kernelSetup(deviceProp);

    /*
    save start of testing in global variable
    make sure it's warmed up before allowing finish loop
     */

    while( badSampleData || std::abs((int)(curTemp - prevTemp)) >= 1) {  
     if (curRun > maxTestRuns ) {
        break;
      }
      printf("  beginning test run %d\n", curRun);
      fflush(stdout);
      setupSampling();
      testClass->runKernel();
      runSampling();

      prevTemp = curTemp;
      curTemp = getDeviceTemp();

     // break; //for profiling runs on kernels

      if (std::abs((int)(curTemp-prevTemp)) >= 1) {
        printf("    temp still changing\n");
      } else {
        badSampleData = !isDataValid();
      }
      curRun++;
    }

    if (maxTestRuns < curRun) {
      printf("maxTestRuns exceeded, sample data may be bad. Debug info:\n");
      printf("  curTemp - prevTemp = %d\n", (int)curTemp-prevTemp);
      printf("  badSampleData: %d\n", badSampleData);
    }
  }

  /*
  analyze test's sampling
    make sure graph is flat and level
    1%-3% tolerance in graph's height
      maybe allow small downward spikes 5ms in width
      very beg and end can ramp up/down though
  return true(good enough) / false(bad samples)
  */
  bool isDataValid() {
    if ((int)powerData.size() < 2*ignoreSampleCount) {
      printf("Only %d samples in previous run\n", (int)powerData.size());
      return false;
    }
    int startPt = ignoreSampleCount;
    int endPt = (int) powerData.size() - ignoreSampleCount;
    double avg = getPowerAvg(startPt, endPt);

    int badStreak = 0;
    double curSample;
    for (int i = startPt; i < endPt; i++) {
      curSample = (double) powerData[i];
      if (std::abs((curSample-avg)/avg) > acceptableError) {

        if (badStreak++ > 250) {
          printf("    power samples not consistant enough. Made it %d%% through data\n", 100*i/endPt);
          return false;
        }

      } else {
        badStreak = 0;
      }
    }

    return true;
  }

  /*
  find the avg power value by random sampling
  */
  double getPowerAvg(int startPt, int endPt) {
    double total = 0;
    for (int i = startPt; i < endPt; i++) {
      total += (double)powerData[i];
    }
    return total / (endPt - startPt);
  }

  /*
  find the avg power value by random sampling for all datapoints
  */
  double getPowerAvg() {
    return getPowerAvg(0, (int) powerData.size() - 1);
  }

  float getElapsedTime() {
    float elapsedT;
    CUDA_ERROR( hipEventElapsedTime(&elapsedT, gpuStart, gpuStop));
    return elapsedT;
  }


  /*
  write data to file
  file is .txt but organized as .csv
    first row: "Power(W), Temperature(ºC), Time(ms), elapsed time(ms), number of samples"
    second row: "xxx, yyy, zzz, aaa, bb"
    third row->end: "xxxx, yyyy, zzzz"
  */
  void dataToFile() {
    float elapsedT;
    CUDA_ERROR( hipEventElapsedTime(&elapsedT, gpuStart, gpuStop));
    printf("  elapsedTime: %f\n", elapsedT);
    float timeBetweenSample = elapsedT / powerData.size();
    
    FILE *fp = fopen(outputName, "w+");
    if (fp == NULL) {
      printf("Attempt at opening '%s' failed. Error: ", outputName);
      perror("");
      printf("Terminating...");
      exit(0);
    }
    fprintf(fp, "Power(W), Temperature(ºC), Time(ms), elapsed time(ms), number of samples, opsPerThread, numberOfThreads\n");
    fprintf(fp, "%.3lf, %d, %f, %3.1f, %d, %d, %d\n", powerData[0]/1000.0, tempData[0], 
            timeBetweenSample, elapsedT, (int)powerData.size(), 
            testClass->getOpsPerThread(), testClass->getNumThreads());
    
    for (int i = 1; i < powerData.size(); i++) {
      fprintf(fp, "%.3lf, %d, %f\n", powerData[i]/1000.0, tempData[0], timeBetweenSample*(i+1) );
    }
    fclose(fp);
  }

  /*
  find number of devices
  */
  void numOfDevices() {
    unsigned int deviceCount;
    nvmlResult = nvmlDeviceGetCount(&deviceCount);
    if (NVML_SUCCESS != nvmlResult)
    {
      printf("Failed to query device count: %s\n", nvmlErrorString(nvmlResult));
      exit(0);
    }
    printf("There are %d GPUs connected, ensure the correct one is selected\n", deviceCount);
  }

  /*
  get device temp
    return temp as unsigned int
    return 0 if there was a sampling error
  */
  unsigned int getDeviceTemp() {
    unsigned int tempLevel;
    nvmlResult = nvmlDeviceGetTemperature(nvmlDeviceID, NVML_TEMPERATURE_GPU, &tempLevel);
    if (nvmlResult != NVML_SUCCESS) {
      tempLevel = 0;
      printf("NVML temp measurement fail: %s\n", nvmlErrorString(nvmlResult));
    }
    return tempLevel;
  }

  unsigned int getPowerUseage() {
    unsigned int powerLevel;
    nvmlResult = nvmlDeviceGetPowerUsage(nvmlDeviceID, &powerLevel);
    if (nvmlResult != NVML_SUCCESS) {
      printf("NVML power measurement fail: %s\n", nvmlErrorString(nvmlResult));
    }
    return powerLevel;
  }

  void CUDA_ERROR(hipError_t e) {
    if (e != hipSuccess) {
      printf("cuda Error: \"%s\"\n", hipGetErrorString(e));
    }
  } 

}; //end TestRunner

